#include "hip/hip_runtime.h"

#include <iostream>

#include <hip/hip_runtime.h>

#include "attractor.cu.h"

__global__ void zero_array(int* array, int array_size) {
	int abs_index = threadIdx.x + blockIdx.x * blockDim.x;
	if (abs_index < array_size)
		array[abs_index] = 0;
}

__global__ void cuda_attractor(int* data, int data_size, int total_thread_count) {
	int threadID = threadIdx.x + blockIdx.x * blockDim.x;

	if (threadID < total_thread_count) {
		std::size_t startPos = (threadID + 1) * 139;
		float_type x = static_cast<float_type>((startPos % img_width * 3 % img_width + margin) % img_width * 4) / img_width - 2,
			y = (static_cast<int>((static_cast<float_type>(startPos) / img_width + 1) * 7) % img_height * 4) / static_cast<float_type>(img_height) - 2; // wrap y value, spread apart, wrap around
	
		int my_iter_count = max(iter_count / total_thread_count, 1u);
	
		for (int i = 0; i < my_iter_count; ++i) {
			float_type x1 = sin(a * y) - cos(b * x),
				y1 = sin(c * x) - cos(d * y);
	
			int x_coord = (x1 + 2) / 4 * (img_width - margin) + margin / 2,
				y_coord = (y1 + 2) / 4 * (img_height - margin) + margin / 2,
				arr_idx = y_coord * img_width + x_coord;

#ifdef ATTRACTOR_ATOMIC_OPS
			++data[arr_idx];
#else
			atomicAdd(data + arr_idx, 1);
#endif
	
			x = x1;
			y = y1;
		}
	}
}

#ifdef __cplusplus
extern "C" {
#endif
	void cuda_zero_array(int array_size, int* array) {
		const int threads_per_block = 512;

		zero_array<<<(array_size + threads_per_block - 1) / threads_per_block, threads_per_block>>>(array, array_size);

		hipDeviceSynchronize();
		hipError_t err = hipGetLastError();

		if (err) {
			std::cerr << "cuda_zero_array failed: " << hipGetErrorString(err) << std::endl;
			std::exit(-1);
		}
	}

	void attractor(int num_threads, int threads_per_block, int* data, int data_size) {
		int num_blocks;
		if (num_threads == -1)
			num_blocks = (data_size + threads_per_block - 1) / threads_per_block;
		else
			num_blocks = (num_threads + threads_per_block - 1) / threads_per_block;

		cuda_attractor<<<num_blocks, threads_per_block>>>(data, data_size, num_threads);

		hipDeviceSynchronize();
		hipError_t err = hipGetLastError();

		if (err) {
			std::cerr << "attractor failed: " << hipGetErrorString(err) << std::endl;
			std::exit(-1);
		}
	}

	int cuda_malloc(void** dst, size_t count) {
		hipError_t err = hipMalloc(dst, count);
		
		if (err) {
			std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << std::endl;
			std::exit(-1);
		}

		return err;
	}

	void cuda_free(void* ptr) {
		hipFree(ptr);
	}

	int cuda_memcpy(void* to, void* from, size_t size, dir_t direction) {
		hipError_t err = hipMemcpy(to, from, size, direction == deviceToHost ? hipMemcpyDeviceToHost : hipMemcpyHostToDevice);

		if (err) {
			std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << std::endl;
			std::exit(-1);
		}

		return err;
	}
#ifdef __cplusplus
}
#endif
